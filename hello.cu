#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define N 1000
#define num_threads 1000

int random_five(int thread, hiprandState state);

int random_seven(int thread, unsigned long long seed);

__global__ void sum_kernel(int *Ad);
__global__ void random_kernel(int *Ad, unsigned long long seed);

int main(void)
{
    int *A;
    int *Ad;
    srand(time(NULL));

    size_t bytes = sizeof(int) * N;

    A = (int*) malloc(bytes);
    hipMalloc(&Ad, bytes);

    random_kernel<<<1,num_threads>>>(Ad, time(NULL));
    hipMemcpy(A, Ad, bytes, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    
    sum_kernel<<<1,7>>>(Ad);

    hipFree(Ad);

    return 0;
}


__global__ void random_kernel(int *Ad, unsigned long long seed)
{
    int i = threadIdx.x;
    int j = blockIdx.x * 1024;
    Ad[i + j] = random_seven(i, seed);
}

__global__ void sum_kernel(int *Ad)
{
    int id = threadIdx.x + 1;
    float result = 0;
    for(int i = 0; i < N; i++)
    {
        if(Ad[i] == id)
            result ++;
    }
    result = (float) result/N * 100;
    printf("Aparaciones del %i es de: %f\n", id, result);
}

__device__ int random_seven(int thread, unsigned long long seed)
{
    hiprandState state;
    hiprand_init(seed, thread, 0, &state);

    int num;
    do {
        int a = random_five(thread, state);
        int b = random_five(thread, state); 
        num = a * 5 + b;     
    } while (num >= 21);     
    printf("%i\n", num % 7 + 1);    
    return num % 7 + 1; 
}

__device__ int random_five(int thread, hiprandState state)
{
    int r;
    int limit = RAND_MAX - (RAND_MAX % 5);
    do {
        r = (hiprand(&state));
    } while (r >= limit);

    return r % 5;
}