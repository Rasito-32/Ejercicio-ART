#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define N 1024000
#define num_threads 1024

int random_five(int thread, int seed);

int random_seven(int thread);

__global__ void sum_kernel(int *Ad);
__global__ void random_kernel(int *Ad);

int main(void)
{
    int *A;
    int *Ad;
    srand(time(NULL));

    size_t bytes = sizeof(int) * N;

    A = (int*) malloc(bytes);
    hipMalloc(&Ad, bytes);

    random_kernel<<<1000,num_threads>>>(Ad);
    hipMemcpy(A, Ad, bytes, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    
    sum_kernel<<<1,7>>>(Ad);

    hipFree(Ad);

    return 0;
}


__global__ void random_kernel(int *Ad)
{
    int i = threadIdx.x;
    int j = blockIdx.x * 1024;
    Ad[i + j] = random_seven(i);
}

__global__ void sum_kernel(int *Ad)
{
    int id = threadIdx.x + 1;
    float result = 0;
    for(int i = 0; i < N; i++)
    {
        if(Ad[i] == id)
            result ++;
    }
    result = (float) result/N * 100;
    printf("Aparaciones del %i es de: %f\n", id, result);
}

__device__ int random_seven(int thread)
{
    int flag = 0;
    int result = 0;
    int count = 0;
    while (flag == 0)
    {
        int num = 5 * random_five(thread, count * 1000) + random_five(thread, count * 1000 - 1);
        if(num < 21)
        {
            result = num % 7 + 1;
            flag = 1;
        }
        count ++;
    }
    return result;
}

__device__ int random_five(int thread, int seed)
{
    hiprandState state;
    hiprand_init(seed, thread, 0, &state); // 1234 is the seed, can be changed
    int random = (hiprand(&state)%5) + 1;

    return random;
}